#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "nvToolsExt.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <chrono>

#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/functional.h>
#include <hip/hip_runtime.h>
#include <thrust/iterator/constant_iterator.h>
#define MAX_DIGITS 32 // Assuming 32-bit integers


using namespace std;


// Anexo A: Radix Sort
// Uno de los algoritmos de ordenamiento más eficientes para ordenar claves cortas en
// procesadores paralelos es el radix sort. El algoritmo comienza considerando el primer bit de
// cada clave, empezando por el bit menos significativo. Utilizando este bit se particiona el
// conjunto de claves de forma que todas las claves que tengan un 0 en ese bit se ubiquen antes
// que las claves que tienen el bit en 1, manteniendo el orden relativo de las claves con mismo
// valor de bit. Una vez completado este paso se hace lo mismo para cada uno de los bits de la
// clave hasta completar todos sus bits.
// Definimos la primitiva split(input, n) como la operación que ordena el arreglo input de acuerdo
// al valor b del bit n de cada elemento. Para implementar en GPU dicha primitiva se procederá
// de la siguiente manera:
// • En un arreglo temporal e almacenar el valor de not b para cada posición i de input.
// • Computar la suma prefija (exclusive scan) del arreglo. Ahora cada posición del arreglo
// contiene la cantidad de f de elementos de input con b=0 que hay antes que esa posición.
// Para los elementos con b=0, esta cantidad determina la posición en el arreglo de salida.
// El último elemento del arreglo de salida del scan contiene el total de posiciones con b=0
// (hay que sumar 1 a este valor si la última posición tiene b=0), denominada totalFalses.
// • Ahora se computa el índice de las posiciones con b=1 en el arreglo de salida. Para cada
// posición i, este índice será t = i - f + totalFalses.
// • Una vez obtenidos los índices anteriores se graba cada elemento de input en el arreglo
// de salida en la posición t o f dependiendo de si b es 1 o 0.
// Para implementar el algoritmo de radix sort utilizando la primitiva split simplemente debe
// inicializarse una máscara binaria para aislar el bit menos significativo, realizar el split del
// arreglo según ese bit, comprobar si el arreglo ya está ordenado y, si no lo está, hacer un shift
// a la izquierda de la máscara y volver a iterar. El procedimiento anterior se ejemplifica en la
// figura.

void radixSort_cpu(std::vector<int>& arr) {
    int n = arr.size();
    if (n == 0) return;

    std::vector<int> output(n);
    std::vector<int> bitArray(n);
    std::vector<int> prefixSum(n);

    for (int bit = 0; bit < MAX_DIGITS; bit++) {
        int mask = 1 << bit;

        // Extract bit
        for (int i = 0; i < n; i++) {
            bitArray[i] = (arr[i] & mask) >> bit;
        }

        // Perform exclusive scan (prefix sum of not bit)
        prefixSum[0] = 0;
        for (int i = 1; i < n; i++) {
            prefixSum[i] = prefixSum[i - 1] + (1 - bitArray[i - 1]);
        }

        int totalFalses = prefixSum[n - 1] + (1 - bitArray[n - 1]);

        // Reorder
        std::fill(output.begin(), output.end(), 0); // Ensure the output vector is cleared

        for (int i = 0; i < n; i++) {
            int destination;
            if (bitArray[i] == 0) {
                destination = prefixSum[i];
            } else {
                destination = i - prefixSum[i] + totalFalses;
            }
            output[destination] = arr[i];
        }

        // Copy back to input array for next iteration
        std::copy(output.begin(), output.end(), arr.begin());
    }
}

void filtro_mediana_cpu(int* img_in, int* img_out, int width, int height, int W) {
    for (int pixel = 0; pixel < width * height; pixel++) {
        int x = pixel % width;
        int y = pixel / width;
        std::vector<int> window;

        for (int i = x - W; i <= x + W; i++) {
            for (int j = y - W; j <= y + W; j++) {
                if (i >= 0 && i < width && j >= 0 && j < height) {
                    window.push_back(img_in[j * width + i]);
                }
            }
        }

        radixSort_cpu(window);
        img_out[pixel] = window[window.size() / 2];
    }
}


 // ...................................................................................................................

#define BITS_PER_PASS 1

// Error checking macro
#define cudaCheckError() {                                      \
    hipError_t e = hipGetLastError();                         \
    if (e != hipSuccess) {                                     \
        printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        exit(1);                                                \
    }                                                           \
}

//--------------------

#define BLOCK_SIZE_RADIX 256


// Función para realizar un split basado en el bit n-ésimo
__device__ void split(int* d_input, int* d_output, int* d_bitArray, int* d_prefixSum, int n, int size) {
    int idx = threadIdx.x;
    if (idx < size) {
        // Extract bit
        int as_int = d_input[idx];
        d_bitArray[idx] = (as_int >> n) & 1;
    }
    __syncthreads();

    // Perform exclusive scan (simplified for shared memory usage)
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        int index = (idx + 1) * stride * 2 - 1;
        if (index < blockDim.x) {
            d_prefixSum[index] += d_prefixSum[index - stride];
        }
        __syncthreads();
    }

    if (idx < size) {
        // Reorder
        int destination;
        if (d_bitArray[idx] == 0) {
            destination = d_prefixSum[idx];
        } else {
            destination = idx - d_prefixSum[idx] + d_prefixSum[size-1] + (d_bitArray[size-1] == 0);
        }
        d_output[destination] = d_input[idx];
    }
    __syncthreads();

    // Copy back to input array for next iteration
    if (idx < size) {
        d_input[idx] = d_output[idx];
    }
}

// Función principal de Radix Sort
__device__ void radixSort(int* d_input, int size) {
    __shared__ int d_output[1024];  // Adjust size as needed, or use dynamic shared memory
    __shared__ int d_bitArray[1024];  // Adjust size as needed
    __shared__ int d_prefixSum[1024]; // Adjust size as needed

    for (int bit = 0; bit < 32; bit++) {
        split(d_input, d_output, d_bitArray, d_prefixSum, bit, size);
    }
}

// Kernel para aplicar el filtro de mediana
__global__ void medianaKernel(int* img_in, int* img_out, int width, int height, int W) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int pixel = y * width + x;

    if (x < width && y < height) {
        __shared__ int window[1024];  // Adjust size as needed, or use dynamic shared memory
        int count = 0;

        for (int i = x - W; i <= x + W; i++) {
            for (int j = y - W; j <= y + W; j++) {
                if (i >= 0 && i < width && j >= 0 && j < height) {
                    window[count++] = img_in[j * width + i];
                }
            }
        }

        radixSort(window, count);

        img_out[pixel] = window[count / 2];
    }
}

void filtro_mediana_gpu(int* img_in, int* img_out, int width, int height, int W) {
    int *d_img_in, *d_img_out;
    size_t size = width * height * sizeof(int);

    hipMalloc(&d_img_in, size);
    hipMalloc(&d_img_out, size);

    hipMemcpy(d_img_in, img_in, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    size_t sharedMemSize = (2 * W + 1) * (2 * W + 1) * sizeof(int);
    medianaKernel<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(d_img_in, d_img_out, width, height, W);
    hipDeviceSynchronize();
    hipMemcpy(img_out, d_img_out, size, hipMemcpyDeviceToHost);

    hipFree(d_img_in);
    hipFree(d_img_out);
}


