#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "nvToolsExt.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <chrono>

#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/functional.h>
#include <hip/hip_runtime.h>
#include <thrust/iterator/constant_iterator.h>

#include <hip/hip_runtime.h>
#include <cub/device/device_radix_sort.cuh>
#include <cub/util_allocator.cuh>
using namespace std;

int getMax_cpu(const std::vector<int>& arr) {
    int max = arr[0];
    for (int num : arr) {
        if (num > max) {
            max = num;
        }
    }
    return max;
}

void countingSort_cpu(std::vector<int>& arr, int exp) {
    int n = arr.size();
    std::vector<int> output(n);
    int count[10] = {0};

    for (int i = 0; i < n; i++) {
        count[(arr[i] / exp) % 10]++;
    }

    for (int i = 1; i < 10; i++) {
        count[i] += count[i - 1];
    }

    for (int i = n - 1; i >= 0; i--) {
        output[count[(arr[i] / exp) % 10] - 1] = arr[i];
        count[(arr[i] / exp) % 10]--;
    }

    for (int i = 0; i < n; i++) {
        arr[i] = output[i];
    }
}

void radixSort_cpu(std::vector<int>& arr) {
    int max = getMax_cpu(arr);

    for (int exp = 1; max / exp > 0; exp *= 10) {
        countingSort_cpu(arr, exp);
    }
}

void filtro_mediana_cpu(int* img_in, int* img_out, int width, int height, int W) {
    for (int pixel = 0; pixel < width * height; pixel++) {
        int x = pixel % width;
        int y = pixel / width;
        int window[(2 * W + 1) * (2 * W + 1)];
        int count = 0;
        for (int i = x - W; i <= x + W; i++) {
            for (int j = y - W; j <= y + W; j++) {
                if (i >= 0 && i < width && j >= 0 && j < height) {
                    window[count++] = img_in[j * width + i];
                }
            }
        }

        // Escalar a enteros
        std::vector<int> int_window(count);
        // for (int k = 0; k < count; k++) {
        //     int_window[k] = static_cast<int>(window[k] * 1000); // Ajustar escala si es necesario
        // }

        // Aplicar Radix Sort
        radixSort_cpu(int_window);

        // Desescalar a flotantes y encontrar la mediana
        // for (int k = 0; k < count; k++) {
        //     window[k] = static_cast<int>(int_window[k]) / 1000.0f; // Ajustar escala si es necesario
        // }

        img_out[pixel] = window[count / 2];
    }
}

 // ...................................................................................................................

#define BITS_PER_PASS 1
#define MAX_DIGITS 32 // Assuming 32-bit integers

// Error checking macro
#define cudaCheckError() {                                      \
    hipError_t e = hipGetLastError();                         \
    if (e != hipSuccess) {                                     \
        printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        exit(1);                                                \
    }                                                           \
}

// Kernel to extract window elements and convert to integers
__global__ void extractWindowKernel(int* img_in, int* windows, int width, int height, int W) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int pixel = y * width + x;
    int windowSize = (2 * W + 1) * (2 * W + 1);

    if (x < width && y < height) {
        int count = 0;
        for (int i = x - W; i <= x + W; i++) {
            for (int j = y - W; j <= y + W; j++) {
                if (i >= 0 && i < width && j >= 0 && j < height) {
                    windows[pixel * windowSize + count] = img_in[j * width + i];
                    count++;
                }
            }
        }
        // Pad the rest of the window with 0
        for (; count < windowSize; count++) {
            windows[pixel * windowSize + count] = 0;  // Max positive int
        }
    }
}

// Kernel to select median and convert back to int
__global__ void selectMedianKernel(int* sortedWindows, int* img_out, int width, int height, int W) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int pixel = y * width + x;
    int windowSize = (2 * W + 1) * (2 * W + 1);

    if (x < width && y < height) {
        img_out[pixel] = sortedWindows[pixel * windowSize + windowSize / 2];
    }
}

void filtro_mediana_gpu(int* img_in, int* img_out, int width, int height, int W) {
    int *d_windows, *d_sortedWindows;
    int *d_img_in, *d_img_out;
    size_t size = width * height * sizeof(int);
    int windowSize = (2 * W + 1) * (2 * W + 1);
    size_t windowsSize = width * height * windowSize * sizeof(int);

    // Allocate device memory
    hipMalloc(&d_img_in, size);
    hipMalloc(&d_img_out, size);
    hipMalloc(&d_windows, windowsSize);
    hipMalloc(&d_sortedWindows, windowsSize);

    // Copy input image to device
    hipMemcpy(d_img_in, img_in, size, hipMemcpyHostToDevice);

    // Extract windows
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    extractWindowKernel<<<blocksPerGrid, threadsPerBlock>>>(d_img_in, d_windows, width, height, W);

    // Sort windows using CUB's DeviceRadixSort
    void* d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    
    // First, get the required temporary storage size
    hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_windows, d_sortedWindows, width * height * windowSize);

    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    // Perform the sort
    hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_windows, d_sortedWindows, width * height * windowSize);

    // Select median
    selectMedianKernel<<<blocksPerGrid, threadsPerBlock>>>(d_sortedWindows, d_img_out, width, height, W);

    // Copy result back to host
    hipMemcpy(img_out, d_img_out, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_img_in);
    hipFree(d_img_out);
    hipFree(d_windows);
    hipFree(d_sortedWindows);
    hipFree(d_temp_storage);
}
