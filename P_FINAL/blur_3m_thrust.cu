#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "nvToolsExt.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <chrono>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/functional.h>
#include <hip/hip_runtime.h>
#include <thrust/iterator/constant_iterator.h>
#include <hip/hip_runtime.h>
#include <thrust/iterator/constant_iterator.h>

using namespace std;

#define MAX_INT 2147483647
struct ExtractWindow {
    int* d_img_in;
    int* d_windows;
    int width;
    int height;
    int W;
    int windowSize;
    
    ExtractWindow(int* _d_img_in, int* _d_windows, int _width, int _height, int _W) 
        : d_img_in(_d_img_in), d_windows(_d_windows), width(_width), height(_height), W(_W) {
        windowSize = (2 * W + 1) * (2 * W + 1);
    }
    
    __device__
    void operator()(int element) const {
        int i = element % windowSize;
        int pixel = element / windowSize;

        int x = pixel % width;

        int y = pixel / width;

        int j = i % (2 * W + 1) - W; 
        int k = i / (2 * W + 1) - W;
    
        int img_x = x + j;
        int img_y = y + k;

        if (img_x >= 0 && img_x < width && img_y >= 0 && img_y < height) {
            d_windows[element] = d_img_in[img_y * width + img_x];
        } else {
            d_windows[element] = 0;
        }
    }
};

struct SelectMedian {
    int* d_windows;
    int* d_img_out;
    int width;
    int height;
    int W;
    int windowSize;
    
    SelectMedian(int* _d_windows, int* _d_img_out, int _width, int _height, int _W) 
        : d_windows(_d_windows), d_img_out(_d_img_out), width(_width), height(_height), W(_W) {
        windowSize = (2 * W + 1) * (2 * W + 1);
    }
    
    __device__
    void operator()(int i) const {
        int x = i % width;
        int y = i / width;
        int pixel = y * width + x;
        d_img_out[pixel] = d_windows[pixel * windowSize + windowSize / 2];
    }
};

struct SortWindow {
    int* d_windows;
    int* d_img_out;
    int windowSize;
    
    SortWindow(int* _d_windows, int* _d_img_out, int _windowSize) 
        : d_windows(_d_windows), d_img_out(_d_img_out), windowSize(_windowSize) {}
    
    __device__
    void operator()(int i) const {
        thrust::sort(thrust::seq, d_windows + i * windowSize, d_windows + (i + 1) * windowSize);
        d_img_out[i] = d_windows[i * windowSize + windowSize / 2];
    }
};

void filtro_mediana_gpu(int* img_in, int* img_out, int width, int height, int W) {
    int *d_windows;
    int *d_img_in, *d_img_out;
    size_t size = width * height * sizeof(int);
    int windowSize = (2 * W + 1) * (2 * W + 1);
    size_t windowsSize = width * height * windowSize * sizeof(int);

    // Allocate device memory
    hipMalloc(&d_img_in, size);
    hipMalloc(&d_img_out, size);
    hipMalloc(&d_windows, windowsSize);

    // Copy input image to device
    hipMemcpy(d_img_in, img_in, size, hipMemcpyHostToDevice);

    thrust::counting_iterator<int> begin(0);
    thrust::counting_iterator<int> end(width * height);
    thrust::counting_iterator<int> endExtractWindow(width * height * windowSize);
    
    thrust::for_each(thrust::device, begin, endExtractWindow, ExtractWindow(d_img_in, d_windows, width, height, W));
    thrust::for_each(thrust::device, begin, end, SortWindow(d_windows, d_img_out, windowSize));

    hipMemcpy(img_out, d_img_out, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_img_in);
    hipFree(d_img_out);
    hipFree(d_windows);
}