#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "nvToolsExt.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <chrono>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/functional.h>
#include <hip/hip_runtime.h>
#include <thrust/iterator/constant_iterator.h>
#include <hip/hip_runtime.h>
#include <thrust/iterator/constant_iterator.h>

using namespace std;

#define MAX_INT 2147483647
// Estructura para extraer la ventana alrededor de cada píxel
struct ExtractWindow {
    int* d_img_in;
    int* d_windows;
    int width;
    int height;
    int W;
    int windowSize;
    
    ExtractWindow(int* _d_img_in, int* _d_windows, int _width, int _height, int _W) 
        : d_img_in(_d_img_in), d_windows(_d_windows), width(_width), height(_height), W(_W) {
        windowSize = (2 * W + 1) * (2 * W + 1);
    }
    
    __device__
    void operator()(int i) const {
        int x = i % width;
        int y = i / width;
        int pixel = y * width + x;
        int count = 0;
        int padding = 0;
        for (int j = y - W; j <= y + W; j++) {
            for (int i = x - W; i <= x + W; i++) {
                if (i >= 0 && i < width && j >= 0 && j < height) {
                    d_windows[pixel * windowSize + count] = d_img_in[j * width + i];
                }
                else {
                    d_windows[pixel * windowSize + count] = padding;
                    padding = MAX_INT-padding; // Para evitar el padding
                }
                count++;
            }
        }
    }
};

// Estructura para seleccionar la mediana de cada ventana
struct SelectMedian {
    int* d_windows;
    int* d_img_out;
    int width;
    int height;
    int W;
    int windowSize;
    
    SelectMedian(int* _d_windows, int* _d_img_out, int _width, int _height, int _W) 
        : d_windows(_d_windows), d_img_out(_d_img_out), width(_width), height(_height), W(_W) {
        windowSize = (2 * W + 1) * (2 * W + 1);
    }
    
    __device__
    void operator()(int i) const {
        int x = i % width;
        int y = i / width;
        int pixel = y * width + x;
        d_img_out[pixel] = d_windows[pixel * windowSize + windowSize / 2];
    }
};


// Estructura para ordenar cada ventana utilizando Thrust
struct SortWindow {
    int* d_windows;
    int windowSize;
    
    SortWindow(int* _d_windows, int _windowSize) 
        : d_windows(_d_windows), windowSize(_windowSize) {}
    
    __device__
    void operator()(int i) const {
        thrust::sort(thrust::seq, d_windows + i * windowSize, d_windows + (i + 1) * windowSize);
    }
};

// Función para aplicar el filtro de mediana en la GPU
void filtro_mediana_gpu(int* img_in, int* img_out, int width, int height, int W) {
    int *d_windows;
    int *d_img_in, *d_img_out;
    size_t size = width * height * sizeof(int);
    int windowSize = (2 * W + 1) * (2 * W + 1);
    size_t windowsSize = width * height * windowSize * sizeof(int);

    // Asigno memoria 
    hipMalloc(&d_img_in, size);
    hipMalloc(&d_img_out, size);
    hipMalloc(&d_windows, windowsSize);

    // Copio la imagen de entrada al dispositivo
    hipMemcpy(d_img_in, img_in, size, hipMemcpyHostToDevice);

    thrust::counting_iterator<int> begin(0);
    thrust::counting_iterator<int> end(width * height);
    
    thrust::for_each(thrust::device, begin, end, ExtractWindow(d_img_in, d_windows, width, height, W));

    // Ordeno cada ventana
    thrust::for_each(thrust::device, begin, end, SortWindow(d_windows, windowSize));

    // selecciono mediana de cada ventana
    thrust::for_each(thrust::device, begin, end, SelectMedian(d_windows, d_img_out, width, height, W));

    // copio al host
    hipMemcpy(img_out, d_img_out, size, hipMemcpyDeviceToHost);

    //libero la memoria
    hipFree(d_img_in);
    hipFree(d_img_out);
    hipFree(d_windows);
}