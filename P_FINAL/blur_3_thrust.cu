#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "nvToolsExt.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <chrono>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/functional.h>
#include <hip/hip_runtime.h>
#include <thrust/iterator/constant_iterator.h>
#include <hip/hip_runtime.h>
#include <thrust/iterator/constant_iterator.h>

using namespace std;

// Kernel to extract window elements and convert to integers
// __global__ void extractWindowKernel(int* img_in, int* windows, int width, int height, int W) {
//     int x = blockIdx.x * blockDim.x + threadIdx.x;
//     int y = blockIdx.y * blockDim.y + threadIdx.y;
//     int pixel = y * width + x;
//     int windowSize = (2 * W + 1) * (2 * W + 1);
//     if (x < width && y < height) {
//         int count = 0;
//         for (int j = y - W; j <= y + W; j++) {
//             for (int i = x - W; i <= x + W; i++) {
//                 if (i >= 0 && i < width && j >= 0 && j < height) {
//                     windows[pixel * windowSize + count] = img_in[j * width + i];
//                 }
//                 else {
//                     windows[pixel * windowSize + count] = 0;
//                 }
//                 count++;
//             }
//         }
//     }
// }
#define MAX_INT 2147483647
struct ExtractWindow {
    int* d_img_in;
    int* d_windows;
    int width;
    int height;
    int W;
    int windowSize;
    
    ExtractWindow(int* _d_img_in, int* _d_windows, int _width, int _height, int _W) 
        : d_img_in(_d_img_in), d_windows(_d_windows), width(_width), height(_height), W(_W) {
        windowSize = (2 * W + 1) * (2 * W + 1);
    }
    
    __device__
    void operator()(int i) const {
        int x = i % width;
        int y = i / width;
        int pixel = y * width + x;
        int count = 0;
        int padding = 0;
        for (int j = y - W; j <= y + W; j++) {
            for (int i = x - W; i <= x + W; i++) {
                if (i >= 0 && i < width && j >= 0 && j < height) {
                    d_windows[pixel * windowSize + count] = d_img_in[j * width + i];
                }
                else {
                    d_windows[pixel * windowSize + count] = padding;
                    padding = MAX_INT-padding; // To avoid sorting the padding
                }
                count++;
            }
        }
    }
};

// // Kernel to select median
// __global__ void selectMedianKernel(int* sortedWindows, int* img_out, int width, int height, int W) {
//     int x = blockIdx.x * blockDim.x + threadIdx.x;
//     int y = blockIdx.y * blockDim.y + threadIdx.y;
//     int pixel = y * width + x;
//     int windowSize = (2 * W + 1) * (2 * W + 1);
//     if (x < width && y < height) {
//         img_out[pixel] = sortedWindows[pixel * windowSize + windowSize / 2];
//     }
// }


struct SelectMedian {
    int* d_windows;
    int* d_img_out;
    int width;
    int height;
    int W;
    int windowSize;
    
    SelectMedian(int* _d_windows, int* _d_img_out, int _width, int _height, int _W) 
        : d_windows(_d_windows), d_img_out(_d_img_out), width(_width), height(_height), W(_W) {
        windowSize = (2 * W + 1) * (2 * W + 1);
    }
    
    __device__
    void operator()(int i) const {
        int x = i % width;
        int y = i / width;
        int pixel = y * width + x;
        d_img_out[pixel] = d_windows[pixel * windowSize + windowSize / 2];
    }
};

struct SortWindow {
    int* d_windows;
    int windowSize;
    
    SortWindow(int* _d_windows, int _windowSize) 
        : d_windows(_d_windows), windowSize(_windowSize) {}
    
    __device__
    void operator()(int i) const {
        thrust::sort(thrust::seq, d_windows + i * windowSize, d_windows + (i + 1) * windowSize);
    }
};

void filtro_mediana_gpu(int* img_in, int* img_out, int width, int height, int W) {
    int *d_windows;
    int *d_img_in, *d_img_out;
    size_t size = width * height * sizeof(int);
    int windowSize = (2 * W + 1) * (2 * W + 1);
    size_t windowsSize = width * height * windowSize * sizeof(int);

    // Allocate device memory
    hipMalloc(&d_img_in, size);
    hipMalloc(&d_img_out, size);
    hipMalloc(&d_windows, windowsSize);

    // Copy input image to device
    hipMemcpy(d_img_in, img_in, size, hipMemcpyHostToDevice);

    thrust::counting_iterator<int> begin(0);
    thrust::counting_iterator<int> end(width * height);
    
    // Extract windows
    thrust::for_each(thrust::device, begin, end, ExtractWindow(d_img_in, d_windows, width, height, W));

    // Sort windows with thrust in parallel
    thrust::for_each(thrust::device, begin, end, SortWindow(d_windows, windowSize));

    // Select median
    thrust::for_each(thrust::device, begin, end, SelectMedian(d_windows, d_img_out, width, height, W));

    // Copy output image to host
    hipMemcpy(img_out, d_img_out, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_img_in);
    hipFree(d_img_out);
    hipFree(d_windows);
}