#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "nvToolsExt.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <chrono>

#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/functional.h>
#include <hip/hip_runtime.h>
#include <thrust/iterator/constant_iterator.h>

using namespace std;


__device__ void swap(float& a, float& b) {
    //swap two elements without using a temporary variable
    a = a + b;
    b = a - b;
    a = a - b;
}

__device__ int partition(float* arr, int low, int high) {
    float pivot = arr[high];
    int i = low - 1;

    for (int j = low; j <= high - 1; j++) {
        if (arr[j] < pivot) {
            i++;
            swap(arr[i], arr[j]);
        }
    }
    swap(arr[i + 1], arr[high]);
    return i + 1;
}

__device__ void quickSort(float* arr, int low, int high) {
    if (low < high) {
        int pi = partition(arr, low, high);
        quickSort(arr, low, pi - 1);
        quickSort(arr, pi + 1, high);
    }
}

__global__ void filtro_mediana_kernel(float* d_input, float* d_output, int width, int height, float W){

    int windowSize = (2 * W + 1) * (2 * W + 1);
    float* window = new float[windowSize];
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int count = 0;

    for (int i = x - W; i <= x + W; i++){
        for (int j = y - W; j <= y + W; j++){
            if (i >= 0 && i < width && j >= 0 && j < height){
                window[count++] = d_input[j * width + i];
            }
        }
    }

    //sort array with a quicksort
    quickSort(window, 0, count - 1);
    d_output[y * width + x] = window[count / 2];

}

void filtro_mediana_gpu(float * img_in, float * img_out, int width, int height, int W){
    std::chrono::high_resolution_clock::time_point start, end;
    start = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < 10; i++){
        float *d_input, *d_output;
        hipMalloc(&d_input, width * height * sizeof(float));
        hipMalloc(&d_output, width * height * sizeof(float));

        hipMemcpy(d_input, img_in, width * height * sizeof(float), hipMemcpyHostToDevice);

        dim3 blockSize(32, 32);
        dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

        filtro_mediana_kernel<<<gridSize, blockSize>>>(d_input, d_output, width, height, W);

        hipMemcpy(img_out, d_output, width * height * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_input);
        hipFree(d_output);    

    }
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> duration = end - start;

    printf("Tiempo GPU: %f\n", duration.count());
    
}

void filtro_mediana_cpu(float * img_in, float * img_out, int width, int height, int W){
    std::chrono::high_resolution_clock::time_point start, end;
    start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < 10; i++){
    //implementar filtro de mediana en CPU 
        for (int pixel = 0; pixel < width * height; pixel++){
            int x = pixel % width;
            int y = pixel / width;
            float window[(2 * W + 1) * (2 * W + 1)];
            int count = 0;
            for (int i = x - W; i <= x + W; i++){
                for (int j = y - W; j <= y + W; j++){
                    if (i >= 0 && i < width && j >= 0 && j < height){
                        window[count++] = img_in[j * width + i];
                    }
                }
            }
            //sort array with a quicksort
            std::sort(window, window + count);
            img_out[pixel] = window[count / 2];
        }
    }
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> duration = end - start;
    printf("Tiempo CPU: %f\n", duration.count());
}
