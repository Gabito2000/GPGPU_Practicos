#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include <iostream>
#include <string>

#define CUDA_CHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void read_file(const char*, int*);
int get_text_length(const char * fname);

#define A 15
#define B 27
#define M 256
#define A_MMI_M -17

#define BLOCK_SIZE 256
#define NUM_BLOCKS 128

__device__ int modulo(int a, int b){
    int r = a % b;
    r = (r < 0) ? r + b : r;
    return r;
}

__global__ void decrypt_kernel(int *d_message, int length)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (int i = idx; i < length; i += stride) {
        d_message[i] = modulo(A_MMI_M * (d_message[i] - B), M);
    }
}

__global__ void count_occurrences_kernel(int *d_message, int length, int *d_occurrences)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (int i = idx; i < length; i += stride) {
        atomicAdd(&d_occurrences[d_message[i]], 1);
    }
}

void print_cuda_error(const std::string &message) {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cout << message << " : " << hipGetErrorString(error) << "\n";
        std::cout << "Error OUT \n";
        exit(1);
    }
}

int main(int argc, char *argv[])
{
    int *h_message;
    int *d_message;
    int *h_occurrences;
    int *d_occurrences;
    unsigned int size;
    const char * fname;

    if (argc < 2) printf("Debe ingresar el nombre del archivo\n");
    else
        fname = argv[1];

    int length = get_text_length(fname);

    size = length * sizeof(int);

    // reservar memoria para el mensaje
    h_message = (int *)malloc(size);

    // leo el archivo de la entrada
    read_file(fname, h_message);

    /* reservar memoria en la GPU */
    CUDA_CHK(hipMalloc((void**)&d_message, size));
    print_cuda_error("Error al reservar memoria en la GPU para el mensaje");

    CUDA_CHK(hipMalloc((void**)&d_occurrences, M * sizeof(int)));
    print_cuda_error("Error al reservar memoria en la GPU para las ocurrencias");

    // Inicializar el vector de ocurrencias con ceros
    CUDA_CHK(hipMemset(d_occurrences, 0, M * sizeof(int)));
    print_cuda_error("Error al inicializar el vector de ocurrencias");

    /* copiar los datos de entrada a la GPU */
    CUDA_CHK(hipMemcpy(d_message, h_message, size, hipMemcpyHostToDevice));
    print_cuda_error("Error al copiar los datos de entrada a la GPU");

    /* Configurar la grilla y lanzar el kernel de desencriptación */
    dim3 blockSize_decrypt(BLOCK_SIZE);
    dim3 numBlocks_decrypt(NUM_BLOCKS);
    
    decrypt_kernel<<<numBlocks_decrypt, blockSize_decrypt>>>(d_message, length);
    print_cuda_error("Error al lanzar el kernel de desencriptación");

    /* Configurar la grilla y lanzar el kernel de conteo de ocurrencias */
    dim3 blockSize_count(BLOCK_SIZE);
    dim3 numBlocks_count(NUM_BLOCKS);
    
    count_occurrences_kernel<<<numBlocks_count, blockSize_count>>>(d_message, length, d_occurrences);
    print_cuda_error("Error al lanzar el kernel de conteo de ocurrencias");

    /* Retornar los datos de las ocurrencias a la CPU */
    h_occurrences = (int *)malloc(M * sizeof(int));
    CUDA_CHK(hipMemcpy(h_occurrences, d_occurrences, M * sizeof(int), hipMemcpyDeviceToHost));
    print_cuda_error("Error al copiar los datos de las ocurrencias a la CPU");

    // Imprimir las ocurrencias de cada caracter
    for (int i = 0; i < M; i++) {
        if (h_occurrences[i] > 0) {
            printf("Caracter '%c': %d ocurrencias\n", (char)i, h_occurrences[i]);
        }
    }

    // Liberar la memoria en la GPU
    CUDA_CHK(hipFree(d_message));
    CUDA_CHK(hipFree(d_occurrences));

    // Liberar la memoria en la CPU
    free(h_message);
    free(h_occurrences);

    return 0;
}

int get_text_length(const char * fname)
{
    FILE *f = NULL;
    f = fopen(fname, "r"); //read and binary flags

    size_t pos = ftell(f);    
    fseek(f, 0, SEEK_END);    
    size_t length = ftell(f); 
    fseek(f, pos, SEEK_SET);  

    fclose(f);

    return length;
}

void read_file(const char * fname, int* input)
{
    FILE *f = NULL;
    f = fopen(fname, "r"); //read and binary flags
    if (f == NULL){
        fprintf(stderr, "Error: Could not find %s file \n", fname);
        exit(1);
    }

    int c; 
    while ((c = getc(f)) != EOF) {
        *(input++) = c;
    }

    fclose(f);
}
