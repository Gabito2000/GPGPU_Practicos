
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 256
#define ITERATIONS 10

__global__ void matrixVectorMultiplication(int *A, int *v, int *x, int numRows) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    for (int j = 0; j < 256; ++j) {
        // x[i] += A[i * 256 + j] * v[j];
        atomicAdd(&x[i], A[i * 256 + j] * v[j]);
    }
}

int main_original() {
    int numRows = 10240;
    int numCols = 256;
    int matrixSize = numRows * numCols;
    int vectorSize = numCols;

    int *h_A = (int*)malloc(matrixSize * sizeof(int));
    int *h_v = (int*)malloc(vectorSize * sizeof(int));
    int *h_x = (int*)malloc(numRows * sizeof(int));

    for (int i = 0; i < matrixSize; ++i) {
        h_A[i] = i;
    }
    for (int i = 0; i < vectorSize; ++i) {
        h_v[i] = i;
    }

    int *d_A, *d_v, *d_x;
    hipMalloc((void**)&d_A, matrixSize * sizeof(int));
    hipMalloc((void**)&d_v, vectorSize * sizeof(int));
    hipMalloc((void**)&d_x, numRows * sizeof(int));

    hipMemcpy(d_A, h_A, matrixSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_v, h_v, vectorSize * sizeof(int), hipMemcpyHostToDevice);

    int gridSize = (numRows + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    
    matrixVectorMultiplication<<<gridSize, BLOCK_SIZE>>>(d_A, d_v, d_x, numRows);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Tiempo de ejecución del kernel: %f ms\n", milliseconds);

    hipMemcpy(h_x, d_x, numRows * sizeof(int), hipMemcpyDeviceToHost);

    free(h_A);
    free(h_v);
    free(h_x);
    hipFree(d_A);
    hipFree(d_v);
    hipFree(d_x);

    // printf("Errors: \n");
    // printf("%s\n", cudaGetErrorString(cudaGetLastError()));

    return 0;
}


int main() {
    for (int i = 0; i < ITERATIONS; i++) {
        main_original();
    }
}