
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 256

#define TILE_WIDTH 16

__global__ void matrixVectorMultiplication(int *A, int *v, int *x, int numRows) {
    __shared__ int shared_v[256];
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;
    
    // Load vector v into shared memory
    if (tx < 256)
        shared_v[tx] = v[tx];
    
    __syncthreads();
    
    // Perform matrix-vector multiplication
    int sum = 0;
    for (int j = 0; j < 256; j += TILE_WIDTH) {
        // Load a tile of the matrix into shared memory
        int tile_start = i * 256 + j;
        int tile_end = min(tile_start + TILE_WIDTH, 256); //min es cuda operation así que no debería de dar porblemas cuando se trabaja con la gpu
        for (int k = tile_start; k < tile_end; ++k) {
            sum += A[k] * shared_v[k - i * 256];
        }
    }
    
    // Accumulate the result
    if (i < numRows)
        atomicAdd(&x[i], sum);
}




int main() {
    // Definir dimensiones de la matriz A y del vector v
    int numRows = 10240;
    int numCols = 256;
    int matrixSize = numRows * numCols;
    int vectorSize = numCols;

    // Reservar memoria en el host
    int *h_A = (int*)malloc(matrixSize * sizeof(int));
    int *h_v = (int*)malloc(vectorSize * sizeof(int));
    int *h_x = (int*)malloc(numRows * sizeof(int));

    // Inicializar matriz A y vector v
    for (int i = 0; i < matrixSize; ++i) {
        h_A[i] = i;
    }
    for (int i = 0; i < vectorSize; ++i) {
        h_v[i] = i;
    }

    // Reservar memoria en el dispositivo
    int *d_A, *d_v, *d_x;
    hipMalloc((void**)&d_A, matrixSize * sizeof(int));
    hipMalloc((void**)&d_v, vectorSize * sizeof(int));
    hipMalloc((void**)&d_x, numRows * sizeof(int));

    // Copiar matriz A y vector v desde el host al dispositivo
    hipMemcpy(d_A, h_A, matrixSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_v, h_v, vectorSize * sizeof(int), hipMemcpyHostToDevice);

    // Definir tamaño de bloque y de la grilla
    int gridSize = (numRows + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Registrar el tiempo de ejecución
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    
    // Ejecutar kernel
    matrixVectorMultiplication<<<gridSize, BLOCK_SIZE>>>(d_A, d_v, d_x, numRows);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("Tiempo de ejecución del kernel: %f ms\n", milliseconds);

    // Copiar vector resultado x desde el dispositivo al host
    hipMemcpy(h_x, d_x, numRows * sizeof(int), hipMemcpyDeviceToHost);

    // Liberar memoria
    free(h_A);
    free(h_v);
    free(h_x);
    hipFree(d_A);
    hipFree(d_v);
    hipFree(d_x);

    //print cuda errors
    // printf("Errors: \n");
    // printf("%s\n", cudaGetErrorString(cudaGetLastError()));

    return 0;
}
