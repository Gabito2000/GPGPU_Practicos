#include "hip/hip_runtime.h"
#include <stdio.h>

#define BLOCK_SIZE 32
#define ITERATIONS 10

__global__ void transposeMatrix(int *inputMatrix, int *outputMatrix, int width, int height) {
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int globalIdy = blockIdx.y * blockDim.y + threadIdx.y;

    if (globalIdx < width && globalIdy < height) {
        int index_input = globalIdy * width + globalIdx;
        int index_output = globalIdx * height + globalIdy;

        outputMatrix[index_output] = inputMatrix[index_input];
    }
}

int main_original() {
    // Define matrix dimensions
    int width = 1024; 
    int height = 1024;
    int matrixSize = width * height;

    // Allocate memory for the matrices on the host
    int *h_inputMatrix = (int*)malloc(matrixSize * sizeof(int));
    int *h_outputMatrix = (int*)malloc(matrixSize * sizeof(int));

    // Initialize input matrix (you can do this randomly or with specific values)
    for (int i = 0; i < matrixSize; ++i) {
        h_inputMatrix[i] = i;
    }

    // Print input matrix
    // printf("Input matrix:\n");
    // for (int i = 0; i < height; ++i) {
    //     for (int j = 0; j < width; ++j) {
    //         printf("%d ", h_inputMatrix[i * width + j]);
    //     }
    //     printf("\n");
    // }

    // Allocate memory for the matrices on the device
    int *d_inputMatrix, *d_outputMatrix;
    hipMalloc((void**)&d_inputMatrix, matrixSize * sizeof(int));
    hipMalloc((void**)&d_outputMatrix, matrixSize * sizeof(int));

    // Copy input matrix from host to device
    hipMemcpy(d_inputMatrix, h_inputMatrix, matrixSize * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Launch kernel
    transposeMatrix<<<gridSize, blockSize>>>(d_inputMatrix, d_outputMatrix, width, height);

    // Copy result back to host
    hipMemcpy(h_outputMatrix, d_outputMatrix, matrixSize * sizeof(int), hipMemcpyDeviceToHost);

    // Print output matrix
    // printf("Output matrix:\n");
    // for (int i = 0; i < width; ++i) {
    //     for (int j = 0; j < height; ++j) {
    //         printf("%d ", h_outputMatrix[i * height + j]);
    //     }
    //     printf("\n");
    // }

    // Free device memory
    hipFree(d_inputMatrix);
    hipFree(d_outputMatrix);

    // Free host memory
    free(h_inputMatrix);
    free(h_outputMatrix);

    return 0;
}

int main() {
    for (int i = 0; i < ITERATIONS; i++) {
        main_original();
}