
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
//uint64_t
#include <stdint.h>

#define BLOCK_SIZE 32
#define ITERATIONS 100


__global__ void transposeMatrix(int *inputMatrix, int *outputMatrix, int width, int height) {
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int globalIdy = blockIdx.y * blockDim.y + threadIdx.y;

    if (globalIdx < width && globalIdy < height) {
        int index_input = globalIdy * width + globalIdx;
        int index_output = globalIdx * height + globalIdy;

        outputMatrix[index_output] = inputMatrix[index_input];
    }
}

uint64_t get_nanoseconds() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ((uint64_t)ts.tv_sec * 1000000000) + ts.tv_nsec;
}


int main() {
    //define block sizes
    int block_sizes[] = {8, 16, 32, 64, 128, 256, 512, 1024};


    // Define matrix dimensions
    int width = 1024; 
    int height = 1024;
    int matrixSize = width * height;
    
    
    // printf("Time for block size, time in ns, ITERATIONS: \n");

    for (int i = 0; i < sizeof(block_sizes) / sizeof(int); i++) {
        int *h_inputMatrix = (int*)malloc(matrixSize * sizeof(int));
        for (int aux = 0; aux < matrixSize; ++aux) {
            h_inputMatrix[aux] = aux;
        }
        
        // declare acc time
        uint64_t acc_time = 0;

        for (int j = 0; j < ITERATIONS; j++) {
            // Allocate memory for the matrices on the host
            int *h_outputMatrix = (int*)malloc(matrixSize * sizeof(int));

            // Allocate memory for the matrices on the device
            int *d_inputMatrix, *d_outputMatrix;
            hipMalloc((void**)&d_inputMatrix, matrixSize * sizeof(int));
            hipMalloc((void**)&d_outputMatrix, matrixSize * sizeof(int));

            // Copy input matrix from host to device
            hipMemcpy(d_inputMatrix, h_inputMatrix, matrixSize * sizeof(int), hipMemcpyHostToDevice);

            // Define grid and block dimensions
            // // dim3 blockSize(BLOCK_SIZE);
            // // dim3 numBlocks(NUM_BLOCKS);
            dim3 blockSize(block_sizes[i]);
            dim3 numBlocks(32);

            // Launch kernel
            uint64_t start = get_nanoseconds();
            transposeMatrix<<<numBlocks, blockSize>>>(d_inputMatrix, d_outputMatrix, width, height);
            uint64_t end = get_nanoseconds();
            acc_time += end - start;
            // Copy result back to host
            hipMemcpy(h_outputMatrix, d_outputMatrix, matrixSize * sizeof(int), hipMemcpyDeviceToHost);

            // Free device memory
            hipFree(d_outputMatrix);

            // Free host memory
            hipFree(d_inputMatrix);
            free(h_outputMatrix);

            hipDeviceReset();
        }

        

        // Free host memory
        free(h_inputMatrix);
        printf("%d, %lu, %d\n", block_sizes[i], (acc_time) / ITERATIONS, ITERATIONS);
    }
    printf("cudaError: %s\n", hipGetErrorString(hipGetLastError()));
    return 0;
}