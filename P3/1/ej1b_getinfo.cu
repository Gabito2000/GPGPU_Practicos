#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


//get the cuda warp size
int getWarpSize() {
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);
    return props.warpSize;
}

int main(){
    printf("Warp size: %d\n", getWarpSize());
    return 0;
}