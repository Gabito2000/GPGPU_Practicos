#include "hip/hip_runtime.h"
// El objetivo del ejercicio es utilizar la memoria compartida para la privatización. Además se utilizará el
// patrón de reducción paralela. Antes de empezar, defina una matriz de enteros de 3840 × 2160 (4K 16:9) y
// adapte el kernel del histograma realizado en el práctico 2 para que trabaje sobre una matriz de enteros entre
// 0 y 255.
// • a) Desarrolle una variante donde cada bloque mantenga un histograma local en memoria compartida
// y, al final de la recorrida de la imagen, los datos del histograma local se impacten en el histograma
// global utilizando atomicAdd.
// • b) Desarrolle otra variante donde, una vez realizada la recorrida de la imagen, cada bloque impacte el
// histograma local en una fila de una “matriz de histogramas”. Luego, otro kernel debe realizar la suma
// por columnas de la matriz para obtener el histograma global.
// – Evite el acceso “no coalesced” eligiendo un tamaño de bloque adecuado.
// – Use el patrón de reducción visto en teórico para sumar cada segmento de una columna.
// – Guarde la suma parcial de cada bloque en la propia matriz de histogramas.
// – Invoque nuevamente el kernel para sumar las sumas parciales del paso anterior hasta que se haya
// sumado toda la columna.
// – Evite acceder a los datos en memoria global de forma “no coalesced” eligiendo un tamaño de
// bloque adecuado.
// – Agregue algunas filas de 0s a la matriz para que el número de filas sea múltiplo del tamaño de
// bloque elegido.
// – Compare el desempeño de esta solución con la de la parte a) y la adaptada del práctico 2.

// Ejercicio 2
// #include <stdio.h>
// #include <stdlib.h>
// #include "hip/hip_runtime.h"
// #include <iostream>
// #include <string>

// #define CUDA_CHK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
// inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
// {
//    if (code != hipSuccess) 
//    {
//       fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
//       if (abort) exit(code);
//    }
// }

// void read_file(const char*, int*);
// int get_text_length(const char * fname);

// #define A 15
// #define B 27
// #define M 256
// #define A_MMI_M -17

// #define BLOCK_SIZE 256
// #define NUM_BLOCKS 128

// __device__ int modulo(int a, int b){
//     int r = a % b;
//     r = (r < 0) ? r + b : r;
//     return r;
// }

// __global__ void decrypt_kernel(int *d_message, int length)
// {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int stride = blockDim.x * gridDim.x;
    
//     for (int i = idx; i < length; i += stride) {
//         d_message[i] = modulo(A_MMI_M * (d_message[i] - B), M);
//     }
// }

// __global__ void count_occurrences_kernel(int *d_message, int length, int *d_occurrences)
// {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int stride = blockDim.x * gridDim.x;
    
//     for (int i = idx; i < length; i += stride) {
//         atomicAdd(&d_occurrences[d_message[i]], 1);
//     }
// }

// void print_cuda_error(const std::string &message) {
//     hipError_t error = hipGetLastError();
//     if (error != hipSuccess) {
//         std::cout << message << " : " << hipGetErrorString(error) << "\n";
//         std::cout << "Error OUT \n";
//         exit(1);
//     }
// }

// int main(int argc, char *argv[])
// {
//     int *h_message;
//     int *d_message;
//     int *h_occurrences;
//     int *d_occurrences;
//     unsigned int size;
//     const char * fname;

//     if (argc < 2) printf("Debe ingresar el nombre del archivo\n");
//     else
//         fname = argv[1];

//     int length = get_text_length(fname);

//     size = length * sizeof(int);

//     // reservar memoria para el mensaje
//     h_message = (int *)malloc(size);

//     // leo el archivo de la entrada
//     read_file(fname, h_message);

//     /* reservar memoria en la GPU */
//     CUDA_CHK(hipMalloc((void**)&d_message, size));
//     print_cuda_error("Error al reservar memoria en la GPU para el mensaje");

//     CUDA_CHK(hipMalloc((void**)&d_occurrences, M * sizeof(int)));
//     print_cuda_error("Error al reservar memoria en la GPU para las ocurrencias");

//     // Inicializar el vector de ocurrencias con ceros
//     CUDA_CHK(hipMemset(d_occurrences, 0, M * sizeof(int)));
//     print_cuda_error("Error al inicializar el vector de ocurrencias");

//     /* copiar los datos de entrada a la GPU */
//     CUDA_CHK(hipMemcpy(d_message, h_message, size, hipMemcpyHostToDevice));
//     print_cuda_error("Error al copiar los datos de entrada a la GPU");

//     /* Configurar la grilla y lanzar el kernel de desencriptación */
//     dim3 blockSize_decrypt(BLOCK_SIZE);
//     dim3 numBlocks_decrypt(NUM_BLOCKS);
    
//     decrypt_kernel<<<numBlocks_decrypt, blockSize_decrypt>>>(d_message, length);
//     print_cuda_error("Error al lanzar el kernel de desencriptación");

//     /* Configurar la grilla y lanzar el kernel de conteo de ocurrencias */
//     dim3 blockSize_count(BLOCK_SIZE);
//     dim3 numBlocks_count(NUM_BLOCKS);
    
//     count_occurrences_kernel<<<numBlocks_count, blockSize_count>>>(d_message, length, d_occurrences);
//     print_cuda_error("Error al lanzar el kernel de conteo de ocurrencias");

//     /* Retornar los datos de las ocurrencias a la CPU */
//     h_occurrences = (int *)malloc(M * sizeof(int));
//     CUDA_CHK(hipMemcpy(h_occurrences, d_occurrences, M * sizeof(int), hipMemcpyDeviceToHost));
//     print_cuda_error("Error al copiar los datos de las ocurrencias a la CPU");

//     // Imprimir las ocurrencias de cada caracter
//     for (int i = 0; i < M; i++) {
//         if (h_occurrences[i] > 0) {
//             printf("Caracter '%c': %d ocurrencias\n", (char)i, h_occurrences[i]);
//         }
//     }

//     // Liberar la memoria en la GPU
//     CUDA_CHK(hipFree(d_message));
//     CUDA_CHK(hipFree(d_occurrences));

//     // Liberar la memoria en la CPU
//     free(h_message);
//     free(h_occurrences);

//     return 0;
// }

// int get_text_length(const char * fname)
// {
//     FILE *f = NULL;
//     f = fopen(fname, "r"); //read and binary flags

//     size_t pos = ftell(f);    
//     fseek(f, 0, SEEK_END);    
//     size_t length = ftell(f); 
//     fseek(f, pos, SEEK_SET);  

//     fclose(f);

//     return length;
// }

// void read_file(const char * fname, int* input)
// {
//     FILE *f = NULL;
//     f = fopen(fname, "r"); //read and binary flags
//     if (f == NULL){
//         fprintf(stderr, "Error: Could not find %s file \n", fname);
//         exit(1);
//     }

//     int c; 
//     while ((c = getc(f)) != EOF) {
//         *(input++) = c;
//     }

//     fclose(f);
// }



#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define WIDTH 3840
#define HEIGHT 2160
#define HISTO_SIZE 256
#define BLOCK_SIZE_x 1024
#define BLOCK_SIZE_y 1

#define ITERATIONS 100

__global__ void histo_kernel_shared(int *d_image, int *d_histogram, int matrixSize) {
    __shared__ int histo_private[HISTO_SIZE];

    //se usa threadIdx.x  ya que habrá 1 en cada posisión de HISTO_SIZE por bloque lo que nos permite asegurarnos de que no sumamos cosas de más y no realizamos operaciones sin sentido

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Inicializar el histograma privado en memoria compartida
    if (threadIdx.x < HISTO_SIZE) {
        histo_private[threadIdx.x] = 0;
    }
    __syncthreads();

    // Calcular el histograma local en memoria compartida
    if (tid < matrixSize) {
        atomicAdd(&histo_private[d_image[tid]], 1);
    }
    
    __syncthreads();

    // Actualizar el histograma global
    if (threadIdx.x < HISTO_SIZE) {
        atomicAdd(&d_histogram[threadIdx.x], histo_private[threadIdx.x]);
    }
}

int main_nuevo() {
    int *h_image = (int *)malloc(WIDTH * HEIGHT * sizeof(int));
    int *d_image, *d_histogram;
    int h_histogram[HISTO_SIZE] = {0};

    // Inicializar imagen con valores aleatorios
    for (int i = 0; i < WIDTH * HEIGHT; i++) {
        h_image[i] = i % HISTO_SIZE;
    }

    hipMalloc((void**)&d_image, WIDTH * HEIGHT * sizeof(int));
    hipMalloc((void**)&d_histogram, HISTO_SIZE * sizeof(int));
    hipMemcpy(d_image, h_image, WIDTH * HEIGHT * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_histogram, 0, HISTO_SIZE * sizeof(int));


    dim3 blockSize(BLOCK_SIZE_x, BLOCK_SIZE_y);
    BLOCK_SIZE = BLOCK_SIZE_x * BLOCK_SIZE_y;
    dim3 numBlocks(WIDTH * HEIGHT + BLOCK_SIZE - 1) / BLOCK_SIZE); //SO it does all the work.



    histo_kernel_shared<<<numBlocks, blockSize>>>(d_image, d_histogram, WIDTH*HEIGHT);

    hipMemcpy(h_histogram, d_histogram, HISTO_SIZE * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < HISTO_SIZE; i++) {
        printf("Bin %d: %d\n", i, h_histogram[i]);
    }

    hipFree(d_image);
    hipFree(d_histogram);
    free(h_image);

    return 0;
}

int main( int argc, char **argv ) {
    for (int i = 0; i < ITERATIONS; i++) {
        main_nuevo();
    }
    return 0;
}
