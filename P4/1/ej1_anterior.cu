
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <stdint.h>

#define ITERATIONS 10
#define WARPSIZE 32

uint64_t get_nanoseconds() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ((uint64_t)ts.tv_sec * 1000000000) + ts.tv_nsec;
}

__global__ void transposeMatrix(int *inputMatrix, int *outputMatrix, int width, int height) {
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int globalIdy = blockIdx.y * blockDim.y + threadIdx.y;

    if (globalIdx < width && globalIdy < height) {
        int index_input = globalIdy * width + globalIdx;
        int index_output = globalIdx * height + globalIdy;

        outputMatrix[index_output] = inputMatrix[index_input];
    }
}

int main_original(int argc, char **argv) {
    int BLOCK_SIZE_x = 32;
    int BLOCK_SIZE_y = 1;

    int width = 1024; 
    int height = 1024;
    int matrixSize = width * height;

    int *h_i = (int*)malloc(matrixSize * sizeof(int));
    int *h_o = (int*)malloc(matrixSize * sizeof(int));

    for (int i = 0; i < matrixSize; ++i) {
        h_i[i] = i;
    }

    // Print input matrix
    // printf("Input matrix:\n");
    // for (int i = 0; i < height; ++i) {
    //     for (int j = 0; j < width; ++j) {
    //         printf("%d ", h_i[i * width + j]);
    //     }
    //     printf("\n");
    // }

    int *d_i, *d_o;
    hipMalloc((void**)&d_i, matrixSize * sizeof(int));
    hipMalloc((void**)&d_o, matrixSize * sizeof(int));

    hipMemcpy(d_i, h_i, matrixSize * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockSize(BLOCK_SIZE_x, BLOCK_SIZE_y);
    dim3 numBlocks(32);

    transposeMatrix<<<numBlocks, blockSize>>>(d_i, d_o, width, height);

    hipMemcpy(h_o, d_o, matrixSize * sizeof(int), hipMemcpyDeviceToHost);

    // Print output matrix
    // printf("Output matrix:\n");
    // for (int i = 0; i < width; ++i) {
    //     for (int j = 0; j < height; ++j) {
    //         printf("%d ", h_o[i * height + j]);
    //     }
    //     printf("\n");
    // }

    hipFree(d_i);
    hipFree(d_o);

    free(h_i);
    free(h_o);

    printf("CUDA ERROR: %s\n", hipGetErrorString(hipGetLastError()));
    return 0;
}

int main(int argc, char **argv) {
    uint64_t start, end;
    start = get_nanoseconds();

    for (int i = 0; i < ITERATIONS; i++) {
        main_original( argc, argv );
        hipDeviceSynchronize();
    }

    end = get_nanoseconds();
    printf("Time: %lu ns\n", end - start);
}